#include <iostream>
#include <chrono>
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include "SieveSortCUDA.cuh"
#include <omp.h>

static void long_test_cuda(const size_t count = 256, const int max_repeats = 1) {
	uint32_t** results_sieve = new uint32_t * [max_repeats];
	uint32_t** results_stdst = new uint32_t * [max_repeats];
//#pragma omp parallel for
	for (int c = 0; c < max_repeats; c++) {
		results_sieve[c] = new uint32_t[count];
		results_stdst[c] = new uint32_t[count];
		for (size_t i = 0; i < count; i++) {
			results_stdst[c][i]
				= results_sieve[c][i]
				= generate_random_32();
		}
	}

	//ok for 16x
	auto start = std::chrono::high_resolution_clock::now();
	for (int c = 0; c < max_repeats; c++) {
		sieve_sort_cuda(results_sieve[c], count);
	}

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed1 = end - start;
	double d1 = ((double)max_repeats / elapsed1.count()) / 1000.0;

	start = std::chrono::high_resolution_clock::now();
	for (int c = 0; c < max_repeats; c++) {
		std::sort(results_stdst[c], results_stdst[c] + count);
	}

	end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed2 = end - start;
	double d2 = ((double)max_repeats / elapsed2.count() / (1000.0));
	std::cout << "==================================" << std::endl;
	//#pragma omp parallel for
	for (int c = 0; c < max_repeats; c++) {
		bool beq = std::equal(results_sieve[c], results_sieve[c] + count, results_stdst[c]);
		if (!beq) {
			for (int d = 0; d < count; d++) {
				if (results_sieve[c][d] != results_stdst[c][d]) {
					std::cout << "found bad value at repeat " << c << " index " << d 
						<<":"<<std::hex<< results_sieve[c][d]<<", "<< results_stdst[c][d]<<std::dec<< std::endl;
				}
			}
		}
		delete[] results_sieve[c];
		delete[] results_stdst[c];
	}
	delete[] results_sieve;
	delete[] results_stdst;

	std::cout << "samples:" << count << std::endl;
	std::cout << "repeats:" << max_repeats << std::endl;
	std::cout << "omp: " << omp_get_max_threads() << " threads" << std::endl;
	std::cout << "sieve sort speed:" << d1 << "K/s" << std::endl;
	std::cout << "std sort speed:  " << d2 << "K/s" << std::endl;
	std::cout << "t1(seive):" << elapsed1.count() << " s" << std::endl;
	std::cout << "t2(std::):" << elapsed2.count() << " s" << std::endl;
	std::cout << "ratio:" << (d1 / d2 * 100.0) << "%" << std::endl;
}
static void long_tests_cuda(size_t start = 12, size_t end = 16) {
	for (size_t i = start; i <= end; i++) {
		std::cout << std::endl;
		std::cout << "i=" << i << std::endl;
		long_test_cuda((1ULL << i), 1);
	}
}

int main()
{
	long_tests_cuda();
	return 0;
}